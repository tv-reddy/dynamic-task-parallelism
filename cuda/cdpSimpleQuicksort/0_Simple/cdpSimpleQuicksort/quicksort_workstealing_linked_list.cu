#include "hip/hip_runtime.h"
/*
* Copyright 1993-2014 NVIDIA Corporation.  All rights reserved.
*
* Please refer to the NVIDIA end user license agreement (EULA) associated
* with this source code for terms and conditions that govern your use of
* this software. Any use, reproduction, disclosure, or distribution of
* this software and related documentation outside the terms of the EULA
* is strictly prohibited.
*
*/
#include <iostream>
#include <cstdio>
#include <hip/hip_runtime_api.h>
#include <helper_string.h>
#include <thrust/device_vector.h>

#define MAX_DEPTH       16
#define MAX_TASKS       6
#define INSERTION_SORT  32

typedef struct Task* Task_t;
typedef struct Deque* Deque_t;
typedef struct Head* Head_t;

struct Task {
    unsigned int* elements;
    unsigned int left;
    unsigned int right;
    unsigned int depth;
};

struct Head {
    unsigned short index;
    unsigned short ctr;
};

struct Deque {
    Head head;
    unsigned int tail;
    Task tasks[MAX_TASKS];    
};

// NULL task
Task nullTask (void) {
    //Task_t taskPtr;
    Task task
    //taskPtr = (Task_t)malloc(sizeof(struct Task));
    task.elements = 0;
    task.left = 0;
    task.right = 0;
    task.depth = 0;
    return task
}

// to push the task on to the work queue
__device__ 
void push(Deque_t queue, Task newTask) 
{   
    if (queue->tail < MAX_TASKS)
    {
        queue->tasks[queue->tail] = newTask;
        queue->tail++;
    }
    else
    {
        // queue->tail = queue->tail % MAX_TASKS;
        // queue->tasks[queue->tail] = newTask;
        // queue->tail++;
        printf("Queue is full!");
    }
}

// pop the task from the work queque
__device__
Task pop(Deque_t queue)
{
    Head oldHead, newHead;
    unsigned int oldTail;
    Task task;

    if(queue->tail == 0)
        return nullTask();

    queue->tail--;
    task = queue->tasks[queue->tail];

    oldHead = queue->head;
    if(queue->tail > oldHead.index)
        return task;
    
    oldTail = queue->tail;
    queue->tail = 0;
    newHead.index = 0;
    newHead.ctr = oldHead.ctr + 1;

    if(oldTail == oldHead.index)
        if(atomicCAS(&(queue->head), oldHead, newHead))
            return task;

    queue->head = newHead;
    return nullTask();
}

// to steal tasks from the work queue
__device__ 
Task steal(Deque_t queue) 
{
    Head oldHead, newHead;
    Task task;

    oldHead = queue->head;
    if(queue->tail <= oldHead.index)
        return nullTask();
    
    task = queue->tasks[oldHead.index];

    newHead = oldHead;
    newHead.index++;
    if( atomicCAS(&(queue->head), oldHead, newHead))
        return task;
    
    // fix this
    return nullTask();
}

////////////////////////////////////////////////////////////////////////////////
// Selection sort used when depth gets too big or the number of elements drops
// below a threshold.
////////////////////////////////////////////////////////////////////////////////
__device__ void selection_sort(unsigned int *data, int left, int right)
{
    for (int i = left ; i <= right ; ++i)
    {
        unsigned min_val = data[i];
        int min_idx = i;

        // Find the smallest value in the range [left, right].
        for (int j = i+1 ; j <= right ; ++j)
        {
            unsigned val_j = data[j];

            if (val_j < min_val)
            {
                min_idx = j;
                min_val = val_j;
            }
        }

        // Swap the values.
        if (i != min_idx)
        {
            data[min_idx] = data[i];
            data[i] = min_val;
        }
    }
}

////////////////////////////////////////////////////////////////////////////////
// Very basic quicksort algorithm, recursively launching the next level.
////////////////////////////////////////////////////////////////////////////////
__global__ void cdp_simple_quicksort(unsigned int *data, int left, int right, int depth)
{
    
    volatile bool lock = True;
    
    while(lock){
        if (blockIdx.x == 0)
        {
            // create deque for this kernel call in global memory
            __global__ Deque_t queue;

            // allocate memory for the deque
            queue = (Deque_t)malloc(sizeof(struct Deque));
            // TODO: Initialize the pointers and tasks array for the queue
            queue->head.index = 0;
            queue->head.ctr = 0;
            queue->tail = 0;
            // If we're too deep or there are few elements left, we use an insertion sort...
            if (depth >= MAX_DEPTH || right-left <= INSERTION_SORT)
            {
                selection_sort(data, left, right);
                return;
            }

            unsigned int *lptr = data+left;
            unsigned int *rptr = data+right;
            unsigned int  pivot = data[(left+right)/2];

            // Do the partitioning.
            while (lptr <= rptr)
            {
                // Find the next left- and right-hand values to swap
                unsigned int lval = *lptr;
                unsigned int rval = *rptr;

                // Move the left pointer as long as the pointed element is smaller than the pivot.
                while (lval < pivot)
                {
                    lptr++;
                    lval = *lptr;
                }

                // Move the right pointer as long as the pointed element is larger than the pivot.
                while (rval > pivot)
                {
                    rptr--;
                    rval = *rptr;
                }

                // If the swap points are valid, do the swap!
                if (lptr <= rptr)
                {
                    *lptr++ = rval;
                    *rptr-- = lval;
                }
            }

            // Now the recursive part
            int nright = rptr - data;
            int nleft  = lptr - data;

            // Launch a new block to sort the left part.
            if (left < (rptr-data))
            {
                // hipStream_t s;
                // hipStreamCreateWithFlags(&s, hipStreamNonBlocking);
                // cdp_simple_quicksort<<< 1, 1, 0, s >>>(data, left, nright, depth+1);
                // hipStreamDestroy(s);

                Task task1;
                task1 = (Task_t)malloc(sizeof(struct Task));
                // TODO: point to the subarray 
                task1->elements = data;
                // TODO: left and right limit
                task1->left = left;
                task1->right = nright;
                // TODO: depth
                task1->depth = depth + 1;

                // TODO: push this task to the queue
                push(queue, task1);
            }

            // Launch a new block to sort the right part.
            if ((lptr-data) < right)
            {
                // hipStream_t s1;
                // hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);
                // cdp_simple_quicksort<<< 1, 1, 0, s1 >>>(data, nleft, right, depth+1);
                // hipStreamDestroy(s1);

                Task task2;
                task2 = (Task_t)malloc(sizeof(struct Task));
                // TODO: point to the subarray 
                task2->elements = data;
                // TODO: left and right limit
                task2->left = nleft;
                task2->right = right;
                // TODO: depth
                task2->depth = depth + 1;
                // TODO: push this task to the queue
                push(queue, task2);
            }

            // the parent block pops the first task
            // TODO: pop the task
            pop(queue);
            // TODO: launch the task
            hipStream_t s;
            hipStreamCreateWithFlags(&s, hipStreamNonBlocking);
            cdp_simple_quicksort<<< 2, 1, 0, s >>>(data, left, nright, depth+1);
            hipStreamDestroy(s);

            // free the lock
            lock = false;
        }

    }

    // second task is stolen by the consumer

    // TODO: steal the task
    Task task = NULL;
    int ATTEMPTS = 10;
    for(int i=0; i < ATTEMPTS; i++) {
        task = steal(queue);
        if (task.elements > 0) {
            break;
        }
    }

    if(task.elements > 0) {
        // TODO: launch the task
        hipStream_t s1;
        hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);
        cdp_simple_quicksort<<< 2, 1, 0, s1 >>>(task.elements, task.left, task.right, task.depth+1);
        hipStreamDestroy(s1);
    } else {
        return;
    }


    // TODO: freeup the memory
    // free(task1);
    // free(task2);
    free(deque);
    
}

////////////////////////////////////////////////////////////////////////////////
// Call the quicksort kernel from the host.
////////////////////////////////////////////////////////////////////////////////
void run_qsort(unsigned int *data, unsigned int nitems)
{
    // Prepare CDP for the max depth 'MAX_DEPTH'.
    checkCudaErrors(hipDeviceSetLimit(cudaLimitDevRuntimeSyncDepth, MAX_DEPTH));

    // Launch on device
    int left = 0;
    int right = nitems-1;
    std::cout << "Launching kernel on the GPU" << std::endl;
    cdp_simple_quicksort<<< 2, 1 >>>(data, left, right, 0);
    checkCudaErrors(hipDeviceSynchronize());
}

////////////////////////////////////////////////////////////////////////////////
// Initialize data on the host.
////////////////////////////////////////////////////////////////////////////////
void initialize_data(unsigned int *dst, unsigned int nitems)
{
    // Fixed seed for illustration
    srand(2047);

    // Fill dst with random values
    for (unsigned i = 0 ; i < nitems ; i++)
        dst[i] = rand() % nitems ;
}

////////////////////////////////////////////////////////////////////////////////
// Verify the results.
////////////////////////////////////////////////////////////////////////////////
void check_results(int n, unsigned int *results_d)
{
    unsigned int *results_h = new unsigned[n];
    checkCudaErrors(hipMemcpy(results_h, results_d, n*sizeof(unsigned), hipMemcpyDeviceToHost));

    for (int i = 1 ; i < n ; ++i)
        if (results_h[i-1] > results_h[i])
        {
            std::cout << "Invalid item[" << i-1 << "]: " << results_h[i-1] << " greater than " << results_h[i] << std::endl;
            exit(EXIT_FAILURE);
        }

    std::cout << "OK" << std::endl;
    delete[] results_h;
}

////////////////////////////////////////////////////////////////////////////////
// Main entry point.
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
    int num_items = 128;
    bool verbose = false;

    if (checkCmdLineFlag(argc, (const char **)argv, "help") ||
        checkCmdLineFlag(argc, (const char **)argv, "h"))
    {
        std::cerr << "Usage: " << argv[0] << " num_items=<num_items>\twhere num_items is the number of items to sort" << std::endl;
        exit(EXIT_SUCCESS);
    }

    if (checkCmdLineFlag(argc, (const char **)argv, "v"))
    {
        verbose = true;
    }

    if (checkCmdLineFlag(argc, (const char **)argv, "num_items"))
    {
        num_items = getCmdLineArgumentInt(argc, (const char **)argv, "num_items");

        if (num_items < 1)
        {
            std::cerr << "ERROR: num_items has to be greater than 1" << std::endl;
            exit(EXIT_FAILURE);
        }
    }

    // Get device properties
    int device_count = 0, device = -1;

    if(checkCmdLineFlag(argc, (const char **)argv, "device"))
    {
        device = getCmdLineArgumentInt(argc, (const char **)argv, "device");

        hipDeviceProp_t properties;
        checkCudaErrors(hipGetDeviceProperties(&properties, device));

        if (properties.major > 3 || (properties.major == 3 && properties.minor >= 5))
        {
            std::cout << "Running on GPU " << device << " (" << properties.name << ")" << std::endl;
        }
        else
        {
            std::cout << "ERROR: cdpsimpleQuicksort requires GPU devices with compute SM 3.5 or higher."<< std::endl;
            std::cout << "Current GPU device has compute SM" << properties.major <<"."<< properties.minor <<". Exiting..." << std::endl;
            exit(EXIT_FAILURE);
        }

    }
    else
    {
        checkCudaErrors(hipGetDeviceCount(&device_count));

        for (int i = 0 ; i < device_count ; ++i)
        {
            hipDeviceProp_t properties;
            checkCudaErrors(hipGetDeviceProperties(&properties, i));

            if (properties.major > 3 || (properties.major == 3 && properties.minor >= 5))
            {
                device = i;
                std::cout << "Running on GPU " << i << " (" << properties.name << ")" << std::endl;
                break;
            }

            std::cout << "GPU " << i << " (" << properties.name << ") does not support CUDA Dynamic Parallelism" << std::endl;
         }
     }

    if (device == -1)
    {
        std::cerr << "cdpSimpleQuicksort requires GPU devices with compute SM 3.5 or higher.  Exiting..." << std::endl;
        exit(EXIT_SUCCESS);
    }

    hipSetDevice(device);

    // Create input data
    unsigned int *h_data = 0;
    unsigned int *d_data = 0;
    
    // Create dummy task
    Task_t h_dummy_task = {0, 0, 0, 0};
    __device__ Task_t d_dummy_task;
    checkCudaErrors(hipMalloc((void **)&d_dummy_task, sizeof(struct Task)));
    checkCudaErrors(hipMemcpy(d_dummy_task, h_dummy_task, sizeof(struct Task), hipMemcpyHostToDevice));

    // Allocate CPU memory and initialize data.
    std::cout << "Initializing data:" << std::endl;
    h_data =(unsigned int *)malloc(num_items*sizeof(unsigned int));
    initialize_data(h_data, num_items);

    if (verbose)
    {
        for (int i=0 ; i<num_items ; i++)
            std::cout << "Data [" << i << "]: " << h_data[i] << std::endl;
    }

    // Allocate GPU memory.
    checkCudaErrors(hipMalloc((void **)&d_data, num_items * sizeof(unsigned int)));
    checkCudaErrors(hipMemcpy(d_data, h_data, num_items * sizeof(unsigned int), hipMemcpyHostToDevice));

    // Execute
    std::cout << "Running quicksort on " << num_items << " elements" << std::endl;
    run_qsort(d_data, num_items);

    // Copy result from GPU back to CPU
    unsigned int *results_h = new unsigned[num_items];
    checkCudaErrors(hipMemcpy(results_h, d_data, num_items*sizeof(unsigned), hipMemcpyDeviceToHost));

    // Check result
    std::cout << "Validating results: ";
    check_results(num_items, d_data);

    // Print result
    std::cout<<"[";
    for(int i = 0; i < num_items; i++) {
        std::cout<<results_h[i];
        if(i < num_items -1) {
            std::cout<<", ";
        }
    }
    std::cout<<"]"<<std::endl;

    free(h_data);
    checkCudaErrors(hipFree(d_data));

    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    hipDeviceReset();
    exit(EXIT_SUCCESS);
}